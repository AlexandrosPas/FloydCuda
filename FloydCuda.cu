#include "hip/hip_runtime.h"
/****************************************************************************
Floyd - Warshall Algorithm developed using CUDA. A 2011-2012 assignement for
Parallel Programming Course of Electrical and Computer Engineering Department
in the Aristotle Faculty of Enginnering - Thessaloniki.

*****************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define INF 100000000
#define BLOCKSIZE 128
#define BITSFT 7 //log2(BLOCKSIZE)


/*****************************************
Array Generator - filling weight matrices
according to Floyd-Warshall theory.
******************************************/
void generate(float *A,float *D,float *Q,int vertices)
{
   int i,j;	
   srand ( time(NULL) );
   for(i=0;i<vertices;i++)
     {
        for(j=0;j<vertices;j++)
	  {
	     A[i*vertices+j]=(float)(rand()%10000);// Insert edge-weight randomly limited to 10000
	     Q[i*vertices+j]=0;			
	     if(i==j)
	       {
		  A[i*vertices+j]=0;
		  D[i*vertices+j]=INF;
	       }
	     else if(A[i*vertices+j]==0)
	       {
		  A[i*vertices+j]=INF;
		  D[i*vertices+j]=INF;
               }
		  if(A[i*vertices+j]!=INF&&D[i*vertices+j]!=INF)
		  D[i*vertices+j]=A[i*vertices+j];
	  }
     }
}

/************************************************************
Serial function of Floyd Warshall Algorithm. Find pseudocode 
at http://en.wikipedia.org/wiki/Floyd_Warshall_algorithm and
translated into C.
*************************************************************/
void sFloyd(float *D,float *Q,int vertices)
{
   int i,j,k;
   for(k=0;k<vertices;k++)
     {
        for(i=0;i<vertices;i++)
	  {
	     for(j=0;j<vertices;j++)
	       {
		  if((D[i*vertices+k]+D[k*vertices+j])<D[i*vertices+j])
		    {
		       D[i*vertices+j]=D[i*vertices+k]+D[k*vertices+j];
		       Q[i*vertices+j]=k;
		    }
	       }
	  }
     }
}

/*****************************************
Parallel Version of Floyd Warshall using
Cuda global.
******************************************/
__global__ void pFloyd(float *D,float *Q,int vertices,int k,int k2)
{
   int i,j,index;
   i= blockIdx.x;
   j=(blockIdx.y << BITSFT) + threadIdx.x;
   index=(i << vertices)+j; 				//vertices equals log2(vertices).
     if((D[(i << vertices)+k]+D[(k2)+j])<D[index])
       {
	  D[index]=D[(i << vertices)+k]+D[(k2)+j];
	  Q[index]=k;
       }
}

/***************************************
Check Function. Check if matrices
D and Q from serial Floyd match the
parallel ones.
****************************************/
void check(float *parallelD,float *D,float *parallelQ,float *Q,int vertices)
{
   int i,j;
   int err=0; 
   for(i=0;i<vertices;i++)
     {
        for(j=0;j<vertices;j++)
	  {
	     printf("parallelD:%f = realD:%f\n",parallelD[i*vertices+j],D[i*vertices+j]);	//Nice old print CHECK
	     if((parallelD[i*vertices+j]!=D[i*vertices+j])||(parallelQ[i*vertices+j]!=Q[i*vertices+j]))
	     err++;
	  }
     }
     printf("ERRORS:%d\n",err);
}	

/*************************************************************/
int main ( int argc, char *argv[] )
{
   float *A, *D, *Q, *parallelD, *parallelQ, *dev_D, *dev_Q;
   int i,vertices,n,k2;	
   if(argc!=2)
   {
      printf("You forgot to ENTER vertices argument.\n./<program name> <number  of vertices>=\n");
      return 1;
   }
   /*Variable init*/	
   vertices=atoi(argv[1]);
   n=(int)log2((float)vertices);
   const int size = vertices*vertices*sizeof(float);
   dim3 dimBlock(BLOCKSIZE,1);
   dim3 dimGrid(vertices,vertices/BLOCKSIZE);
   hipMalloc( (void**)&dev_D, size);
   hipMalloc( (void**)&dev_Q, size);
   A=(float*)malloc(size);
   D=(float*)malloc(size);	
   Q=(float*)malloc(size);
   parallelD=(float*)malloc(size);
   parallelQ=(float*)malloc(size);
   generate(A,D,Q,vertices);

   struct timeval first, second, lapsed;
   struct timezone tzp;
   gettimeofday(&first, &tzp); //Calculation time plus GPU memory transfer time.
   hipMemcpy(dev_D,D,size,hipMemcpyHostToDevice);
   hipMemcpy(dev_Q,Q,size,hipMemcpyHostToDevice);
//	gettimeofday(&first,&tzp);  //calculation's duration time ONLY.
   for(i=0;i<vertices;i++)
     {	
        k2=i*vertices;	
	pFloyd<<<dimGrid,dimBlock>>>(dev_D,dev_Q,n,i,k2);
     }
   hipDeviceSynchronize();
//	gettimeofday(&second,&tzp);  //calculation's duration time ONLY.
   hipMemcpy(parallelD,dev_D,size,hipMemcpyDeviceToHost);
   hipMemcpy(parallelQ,dev_Q,size,hipMemcpyDeviceToHost);
   sFloyd(D,Q,vertices);
   gettimeofday(&second, &tzp);  //Calculation time plus GPU memory transfer time.
   if(first.tv_usec>second.tv_usec)
     {
        second.tv_usec += 1000000;
        second.tv_sec--;
     }
   lapsed.tv_usec = second.tv_usec - first.tv_usec;
   lapsed.tv_sec = second.tv_sec - first.tv_sec;
   check(parallelD,D,parallelQ,Q,vertices);
   printf("Time elapsed: %lu, %lu s\n", lapsed.tv_sec,lapsed.tv_usec);
   hipFree(dev_D);
   hipFree(dev_Q);
   free(A);
   free(D);
   free(Q);
   free(parallelD);
   free(parallelQ);
   return 0;
}
